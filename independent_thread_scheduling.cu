
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define checkCudaErrors(call)                                 \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)

int mutex = 0;

__device__ long getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

__global__ void demo(dim3* threads, dim3* blocks, volatile int* mutex) {
    // locked = 1, unlocked = 0
    printf("here\n");
    while(atomicCAS((unsigned int*) mutex, (unsigned int) 0, (unsigned int) 1) == 1);
    long tid = getIdx(threads, blocks);
    printf("TID: %ld\n", tid);

    *mutex = (unsigned int) 0;
}

void demo_setup(dim3 threadsPerBlock, dim3 blocksPerGrid) {
    dim3* d_threads;
    dim3* d_blocks;
    checkCudaErrors(hipMalloc((void**) &d_threads,
        sizeof(dim3)));
    checkCudaErrors(hipMalloc((void**) &d_blocks,
        sizeof(dim3)));
    checkCudaErrors(hipMemcpy(d_threads, &threadsPerBlock,
        sizeof(dim3), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_blocks, &blocksPerGrid,
        sizeof(dim3),hipMemcpyHostToDevice));
    
    int* m;
    checkCudaErrors(hipMalloc((int**) &m, sizeof(int)));
    checkCudaErrors(hipMemcpy(m, &mutex, sizeof(int),
        hipMemcpyHostToDevice));
    demo<<<blocksPerGrid, threadsPerBlock>>>(d_threads, d_blocks, m);
    checkCudaErrors(hipMemcpy(&mutex, m, sizeof(int),
        hipMemcpyDeviceToHost));
}

int main(int argc, char *argv[]) {
    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 1;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    printf("before demo\n");
    demo_setup(threadsPerBlock, blocksPerGrid);
    printf("after demo\n");
}